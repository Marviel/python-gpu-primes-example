
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>
#include <ctime>
#include <fstream>
#include <cmath>
#include <cstdlib>

using namespace std;

__global__ static void Init(char* primes)
{
   primes[0] = 1;
   primes[1] = 1;
}

__global__ static void sieveEvensCUDA(char* primes, int max)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x + threadIdx.x + 4;
	if (index < max) {
		primes[index] = 1;
	}
}

__global__ static void sieveCUDA(char *primes, int max, int root)
{
  
   int index = blockIdx.x * blockDim.x + threadIdx.x + 3;

   if (index < root && primes[index] == 0)
   {	  
      for (int j = index * index; j <= max; j += index)
      {
         primes[j] = 1;
      }
   }
}

__host__ void isPrime(char* primes, int max)
{
	int blockSize = 32;
	int root = sqrt(max);
	char* d_Primes = NULL;
   
	int sizePrimes = sizeof(char) * max;
   
	hipMalloc(&d_Primes, sizePrimes);
	hipMemset(d_Primes, 0, sizePrimes);
   
	dim3 dimBlock(blockSize);
	dim3 dimGrid((root + dimBlock.x) / dimBlock.x);
	dim3 dimGridEven((max + dimBlock.x) / dimBlock.x);
   
	Init<<<1,1>>>(d_Primes);
	sieveEvensCUDA<<<dimGridEven, dimBlock>>>(d_Primes, max);
	sieveCUDA<<<dimGrid, dimBlock>>>(d_Primes, max, root);
   
	hipMemcpy(primes, d_Primes, sizePrimes, hipMemcpyDeviceToHost);
   
	hipFree(d_Primes);
   
}

int main(){

	int num;
	struct timeval start, end;
    long mtime, seconds, useconds;
	long long sum;    
	char *primes;

	cout << "Enter max positive number to sum primes to: " << endl;
	cin >> num;
	
	primes = (char*)malloc(num);
	memset(primes, 0, num);
	
	if (num < 2) {
		cout << "Entered number less than 2, no primes to sum" << endl;;
	}
	else 
		sum = 2;
	
	gettimeofday(&start, NULL);

	isPrime(primes, num);
	
	for (int n = 3; n <= num - 1; n += 2) {
		if (primes[n] == 0){
			sum += n;
			if(num >= 1 + n*n && num < (n+1)*(n + 1)) {
				sum -= n*n;
			}
		}
	}
	
	gettimeofday(&end, NULL);

	seconds  = end.tv_sec  - start.tv_sec;
	useconds = end.tv_usec - start.tv_usec;

	mtime = ((seconds) * 1000 + useconds/1000.0);

	cout << "Sum of prime numbers under " << num << " is " << sum << endl;
	cout << "Total elapsed time: " << mtime << " milliseconds\n" << endl;
	
	free(primes);
	
	return 0;
}