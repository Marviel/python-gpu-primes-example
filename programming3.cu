
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>
#include <ctime>
#include <fstream>
#include <cmath>
#include <cstdlib>

using namespace std;


//Eratosthanes' sieve on odds
__global__ static void sieve(char *primes, int n, int root)
{
  
   int i = blockIdx.x * blockDim.x + threadIdx.x + 3;

   if (i < root && primes[i] == 0)
   {	  
      for (long j = i * i; j <= n; j += i)
      {
         primes[j] = 1;
      }
   }
}

//Eratosthanes' sieve on evens
__global__ static void Evens(char* P, int n)
{
	long i = blockIdx.x * blockDim.x + threadIdx.x + threadIdx.x + 4;
	if (i < n) {
		P[i] = 1;
	}
}

__global__ static void Init(char* P)
{
   P[0] = 1;
   P[1] = 1;
}

__host__ void isPrime(char* P, int max)
{
	int blockSize = 32;
	long root = sqrt(max);
	char* d_Primes = NULL;
   
	long sizePrimes = sizeof(char) * max;
   
	hipMalloc(&d_Primes, sizePrimes);
	hipMemset(d_Primes, 0, sizePrimes);
   
	dim3 dimBlock(blockSize);
	dim3 dimGrid((root + dimBlock.x) / dimBlock.x);
	dim3 dimGridEven((max + dimBlock.x) / dimBlock.x);
   
	Init<<<1,1>>>(d_Primes);
	Evens<<<dimGridEven, dimBlock>>>(d_Primes, max);
	sieve<<<dimGrid, dimBlock>>>(d_Primes, max, root);
   
	hipMemcpy(P, d_Primes, sizePrimes, hipMemcpyDeviceToHost);
   
	hipFree(d_Primes);
   
}



int main(){
  struct timeval start, end;
  long mtime, seconds, useconds;
	char *primes;

	long long sum;    
	long long num;
	
	cout << "enter number to sum primes to: " << endl;
	cin >> num;
	
	primes = (char*)malloc(num);
	memset(primes, 0, num);
	
	if (num < 2) {
		cout << "no primes to sum!" << endl;;
		return 0;
	}
	else{ 
		sum = 2;
	}

	gettimeofday(&start, NULL);

	isPrime(primes, num);
	
	for (long n = 3; n <= num - 1; n += 2) {
		if (primes[n] == 0){ //Indicates primacy
			//cout << n << " is prime." << endl;
			sum += n;
			if(num >= 1 + n*n && num < (n+1)*(n + 1)) {
				sum -= n*n;
			}
		}
	}

	free(primes);


	gettimeofday(&end, NULL);
	seconds  = end.tv_sec  - start.tv_sec;
	useconds = end.tv_usec - start.tv_usec;
	mtime = ((seconds) * 1000 + useconds/1000.0);

	cout << "sum under " << num << " is " << sum << endl;	
	cout << "time: " << mtime << " milliseconds\n" << endl;
	
	
	return 0;
}
