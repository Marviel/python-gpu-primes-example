#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  y[i] = (float*)i%n;

  // f (i%n != 0) {
  //   y[i] = 1;
  // }
  // else{
  //   y[i] = 0;
  // }
}

int main(void)
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  for(i=0; i < n; i ++)
  {
    P = 1;

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(i, 2.0f, d_x, d_y);
    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    // float maxError = 0.0f;
    // for (int i = 0; i < N; i++)
    //   printf("Max error: %fn", maxError);


    for(j = 2; j < i; j ++)
    {
      if(y[j] == 0)
      {
        P = 0;
        break;
      }
    }
    if(P == 1)
    {
      printf("Prime: %f"%(i))
      sum = sum + i;
    }
  }
  cout << sum << endl;
}