#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, int a, int *x, int *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  y[i] = i%n;

  // f (i%n != 0) {
  //   y[i] = 1;
  // }
  // else{
  //   y[i] = 0;
  // }
}

int main(void)
{
  int N = 1<<20;
  int sum;
  int *x, *y, *d_x, *d_y;
  int P, n;

  n = 1000;

  // printf("Please enter threshold for prime number summation\n");
  // puts(n);

  x = (int*)malloc(N*sizeof(int));
  y = (int*)malloc(N*sizeof(int));

  hipMalloc(&d_x, N*sizeof(int)); 
  hipMalloc(&d_y, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    x[i] = 1;
    y[i] = 2;
  }

  hipMemcpy(d_x, x, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(int), hipMemcpyHostToDevice);

  for(int i=0; i < n; i ++)
  {
    P = 1;

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(i, 2.0f, d_x, d_y);
    hipMemcpy(y, d_y, N*sizeof(int), hipMemcpyDeviceToHost);

    // float maxError = 0.0f;
    // for (int i = 0; i < N; i++)
    //   printf("Max error: %fn", maxError);


    printf("y==============\n")
    for(int j = 2; j < i; j ++)
    {
      printf("%d, ",y[j])
      if(y[j] == 0)
      {
        P = 0;
        break;
      }
    }
    printf("\n")
    if(P == 1)
    {
      printf("Prime: %d  ",i);
      sum = sum + i;
    }
  }
  //cout << sum << endl;
}