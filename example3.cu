
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void saxpy(int n, int a, int *x, int *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  //y[i] = i%n;
	y[i] = 10;

  // f (i%n != 0) {
  //   y[i] = 1;
  // }
  // else{
  //   y[i] = 0;
  // }
}

int main(void)
{
  //int N = 1<<20;
  int sum;
  int *x, *y, *d_x, *d_y;
  int P, n;

  n = 1000;

  // printf("Please enter threshold for prime number summation\n");
  // puts(n);

  x = (int*)malloc(n*sizeof(int));
  y = (int*)malloc(n*sizeof(int));

  hipMalloc(&d_x,n*sizeof(int)); 
  hipMalloc(&d_y,n*sizeof(int));

  for (int i = 0; i < n; i++) {
    x[i] = 1;
    y[i] = 1;
  }

  
  for(int i=0; i < n; i ++)
  {
    P = 1;
		hipMemcpy(d_x, x,n*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_y, y,n*sizeof(int), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    saxpy<<<n,1>>>(i, 2.0f, d_x, d_y);
    hipMemcpy(y, d_y,n*sizeof(int), hipMemcpyDeviceToHost);


    printf("y==============\n");
    for(int j = 2; j < i; j ++)
    {
      printf("%d, ",y[j]);
      if(y[j] == 0)
      {
        P = 0;
        break;
      }
    }
    printf("\n");
    if(P == 1)
    {
      printf("Prime: %d  ",i);
      sum = sum + i;
    }
  }
}
