#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, int a, int *x, int *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  y[i] = i%n;

  // f (i%n != 0) {
  //   y[i] = 1;
  // }
  // else{
  //   y[i] = 0;
  // }
}

int main(void)
{
  int N = 1<<20;
  int sum;
  int *x, *y, *d_x, *d_y;
  x = (int*)malloc(N*sizeof(int));
  y = (int*)malloc(N*sizeof(int));

  hipMalloc(&d_x, N*sizeof(int)); 
  hipMalloc(&d_y, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    x[i] = 1;
    y[i] = 2;
  }

  hipMemcpy(d_x, x, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(int), hipMemcpyHostToDevice);

  for(i=0; i < n; i ++)
  {
    P = 1;

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(i, 2.0f, d_x, d_y);
    hipMemcpy(y, d_y, N*sizeof(int), hipMemcpyDeviceToHost);

    // float maxError = 0.0f;
    // for (int i = 0; i < N; i++)
    //   printf("Max error: %fn", maxError);


    for(j = 2; j < i; j ++)
    {
      if(y[j] == 0)
      {
        P = 0;
        break;
      }
    }
    if(P == 1)
    {
      printf("Prime: %f"%(i))
      sum = sum + i;
    }
  }
  //cout << sum << endl;
}