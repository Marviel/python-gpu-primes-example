#include "hip/hip_runtime.h"
int* d_C;

using namespace std;

__global__ void primo(int* C, int N, int multi)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < N) 
  {
    if(i%2==0||i%3==0||i%5==0||i%7==0)
    {
      C[i]=0;           
    }
    else
    {
      C[i]=i+N*multi;
    }
  }
}

int main()
{
  cout<<"Prime numbers \n";
  int N=1000;
  int h_C[1000];
  size_t size=N* sizeof(int);
  hipMalloc((void**)&d_C, size);

  int threadsPerBlock = 1024;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  vector<int> lista(100000000);
  int c_z=0;

  for(int i=0;i<100000;i++)
  {
    primo<<<blocksPerGrid, threadsPerBlock>>>(d_C, N,i);    
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);         
    for(int c=0;c<N;c++)
    {   
      if(h_C[c]!=0)
      {
        lista[c+N*i-c_z]=h_C[c];
      }
      else
      {
        c_z++;
      }
    }   
  }
  lista.resize(lista.size()-c_z+1);
  return(0);
}